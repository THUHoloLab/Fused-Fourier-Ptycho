#include "hip/hip_runtime.h"
#include "helpers.cuh"

void FFT_sample_forward(
    const int sz_h,
    const int sz_w,
    const creal32_T* __restrict__ x,
    creal32_T* __restrict__ y
){
    dim3 N_THREADS(BLOCK_X,BLOCK_Y,1);
    dim3 N_BLOCKS = {
        (unsigned) ((sz_h + BLOCK_X - 1) / BLOCK_X),
        (unsigned) ((sz_w + BLOCK_Y - 1) / BLOCK_Y),
        (unsigned) 1
    };

    hipfftHandle plan;
    hipfftPlan2d(&plan, sz_h, sz_w, HIPFFT_C2C);
    hipfftExecC2C(plan, (hipfftComplex *)&x[0], (hipfftComplex *)&y[0],HIPFFT_FORWARD);
    hipfftDestroy(plan);

    cufftShift_2D_kernel<<<N_BLOCKS, N_THREADS>>>(y,sz_h);
}

void FFT_sample_backward(
    const dim3 imgSz,
    creal32_T* __restrict__ y
){
    dim3 N_THREADS(BLOCK_X,BLOCK_Y,1);
    dim3 N_BLOCKS = {
        (unsigned) ((imgSz.x + BLOCK_X - 1) / BLOCK_X),
        (unsigned) ((imgSz.y + BLOCK_Y - 1) / BLOCK_Y),
        (unsigned) 1
    };
    cufftShift_2D_kernel<<<N_BLOCKS, N_THREADS>>>(y,(int) imgSz.x);

    hipfftHandle plan;
    hipfftPlan2d(&plan, imgSz.x, imgSz.y, HIPFFT_C2C);
    hipfftExecC2C(plan, (hipfftComplex *)&y[0], (hipfftComplex *)&y[0],HIPFFT_BACKWARD);
    hipfftDestroy(plan);

    ifftCorrection<<<N_BLOCKS, N_THREADS>>>(y,imgSz);
}

void getLatentZ(
    const unsigned z_offset,
    const creal32_T* __restrict__ wavefront1,
    const creal32_T* __restrict__ pupil,
    const int2* __restrict__ ledIdx,
    const dim3 N_BLOCKS,
    const dim3 imLs_sz,
    const dim3 imHs_sz,
    creal32_T* __restrict__ supwave,
    creal32_T* __restrict__ latentz
){
    
    dim3 N_THREADS(BLOCK_X,BLOCK_Y,1);

    int inembed[2];
    for (int i{0}; i < 2; i++) {
        inembed[i] = (int) imLs_sz.x;
    }

    getSubpupil<<<N_BLOCKS, N_THREADS>>>(
        z_offset,
        wavefront1, 
        pupil,
        ledIdx,
        imLs_sz,
        imHs_sz,
        supwave,
        latentz
    );

    cufftShift_2D_kernel<<<N_BLOCKS, N_THREADS>>>(latentz, (unsigned) imLs_sz.x);

    hipfftHandle plan;
    hipfftPlanMany(
        &plan, 
        2, 
        &inembed[0], // n
        &inembed[0], // inembed
        1,           // istride
        (int) imLs_sz.x * (int) imLs_sz.y, // idist
        &inembed[0], // inembed
        1,           // istride
        (int) imLs_sz.x * (int) imLs_sz.y, // idist
        HIPFFT_C2C, 
        (int) imLs_sz.z
    );
    hipfftExecC2C(plan, (hipfftComplex *)&latentz[0], (hipfftComplex *)&latentz[0], HIPFFT_BACKWARD);
    hipfftDestroy(plan);
}



void backwardLatentZ(
    const creal32_T* __restrict__ latentz,
    const dim3 imLs_sz,
    creal32_T* __restrict__ latentz_record
){
    int inembed[2];
    for (int i{0}; i < 2; i++) {
        inembed[i] = (int) imLs_sz.x;
    }

    dim3 N_BLOCKS = {
        (unsigned) ((imLs_sz.x + BLOCK_X - 1) / BLOCK_X),
        (unsigned) ((imLs_sz.y + BLOCK_Y - 1) / BLOCK_Y),
        (unsigned) imLs_sz.z
    };
    dim3 N_THREADS(BLOCK_X, BLOCK_Y, 1);

    hipfftHandle plan;
    hipfftPlanMany(
        &plan, 
        2, 
        &inembed[0], // n
        &inembed[0], // inembed
        1,           // istride
        (int) imLs_sz.x * (int) imLs_sz.y, // idist
        &inembed[0], // inembed
        1,           // istride
        (int) imLs_sz.x * (int) imLs_sz.y, // idist
        HIPFFT_C2C, 
        (int) imLs_sz.z
    );
    hipfftExecC2C(plan, (hipfftComplex *)&latentz[0], (hipfftComplex *)&latentz_record[0], HIPFFT_FORWARD);
    hipfftDestroy(plan);

    cufftShift_2D_kernel<<<N_BLOCKS, N_THREADS>>>(latentz_record, (unsigned) imLs_sz.x);
}

// void update_forward(
//     const creal32_T *wavefront1,
//     const creal32_T *wavefront2,
//     const dim3 imLs_sz,
//     const dim3 imHs_sz,
//     const real32_T * d_obseY,
//     const int2 *d_ledIdx,
//     creal32_T * dldw1,
//     creal32_T * dldw2
// ){
//     // allocate temp variables
//     creal32_T * __restrict__ latentZ;
//     creal32_T * __restrict__ subwave;
//     creal32_T * __restrict__ recordZ;
//     real32_T * __restrict__ dodx;
//     real32_T * __restrict__ dody;
//     real32_T * __restrict__ absO;

//     unsigned arraysize = imLs_sz.x * imLs_sz.y * imLs_sz.z;
//     hipMalloc((creal32_T**)&latentZ, (2*arraysize) * sizeof(float));
//     hipMalloc((creal32_T**)&subwave, (2*arraysize) * sizeof(float));
//     hipMalloc((creal32_T**)&recordZ, (2*arraysize) * sizeof(float));

//     hipMalloc((real32_T**)&dodx, (arraysize) * sizeof(float));
//     hipMalloc((real32_T**)&dody, (arraysize) * sizeof(float));
//     hipMalloc((real32_T**)&absO, (arraysize) * sizeof(float));

//     dim3 N_BLOCKS = {
//         (unsigned) ((imLs_sz.x + BLOCK_X - 1) / BLOCK_X),
//         (unsigned) ((imLs_sz.y + BLOCK_Y - 1) / BLOCK_Y),
//         (unsigned) imLs_sz.z
//     };

//     dim3 N_BLOCKS_L = {
//         (unsigned) ((imHs_sz.x + BLOCK_X - 1) / BLOCK_X),
//         (unsigned) ((imHs_sz.y + BLOCK_Y - 1) / BLOCK_Y),
//         (unsigned) 1
//     };

//     dim3 N_THREADS(BLOCK_X,BLOCK_Y,1);

//     FFT_sample_forward(imHs_sz.x,imHs_sz.y,wavefront1,dldw1);
    
//     getLatentZ(dldw1,wavefront2,d_ledIdx,
//                 imLs_sz,imHs_sz,
//                 subwave,latentZ);

//     getAmplitude<<<N_BLOCKS, N_THREADS>>>(latentZ, imLs_sz, pratio, absO);

//     getGradients<<<N_BLOCKS, N_THREADS>>>(absO, d_obseY, imLs_sz, dodx, dody);

//     differenceMap<<<N_BLOCKS, N_THREADS>>>(dodx,dody,imLs_sz,absO);

//     newX<<<N_BLOCKS, N_THREADS>>>(absO, imLs_sz, pratio, latentZ);
    
//     backwardLatentZ(latentZ, imLs_sz, recordZ);

//     deconvPIE<<<N_BLOCKS, N_THREADS>>>(recordZ, wavefront2, imLs_sz, latentZ);

//     ReduceAddpupil<<<N_BLOCKS, N_THREADS>>>(recordZ, subwave, imLs_sz, dldw2);
    
//     clear_spectrum<<<N_BLOCKS_L, N_THREADS>>>(imHs_sz, dldw1);

//     stitch_spectrum<<<N_BLOCKS, N_THREADS>>>(latentZ, d_ledIdx, imLs_sz, imHs_sz, dldw1);

//     FFT_sample_backward(imHs_sz, dldw1);


//     hipFree(latentZ);
//     hipFree(subwave);
//     hipFree(recordZ);
//     hipFree(dodx);
//     hipFree(dody);
//     hipFree(absO);
// }
